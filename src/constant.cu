int main( void ) {
    // capture the start time
    hipEvent_t start;
    hipEvent_t stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

	CPUBitmap bitmap ( DIM, DIM);
	unsigned char *dev_bitmap;

	// allocate memory on teh GPU for the output bitmap
	HANDLE_ERROR( hipMalloc( (void**) &dev_bitmap, bitmap.image_size() ) );
	
	Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
	for (int i=0; i<SPHERES; i++) {
		temp_s[i].r = rnd( 1.0f );
		temp_s[i].g = rnd( 1.0f );
		temp_s[i].b = rnd( 1.0f );
		temp_s[i].x = rnd( 1000.0f ) - 500;
		temp_s[i].x = rnd( 1000.0f ) - 500;
		temp_s[i].x = rnd( 1000.0f ) - 500;
		temp_s[i].radius = rnd( 100.0f ) + 20;
	}
	handle_error( hipMemcpyToSymbol(HIP_SYMBOL( s), temp_s, sizeof(Sphere) * SPHERES) );
	free( temp_s);

	// generate a bitmap from our sphere data
	dim3 grids(DIM/16,DIM/16);
	dim3 threads(16,16);

