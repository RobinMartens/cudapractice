
#include <hip/hip_runtime.h>
// this is dull!
__global__ void vector_add(int *a, int *b, int *out, int max) {
	int tid = blockIdx.x; 
	if(tid < max) {
		out[tid] = a[tid] + b[tid];
	}
}

__global__ void intmap(int *a, int *out, int (*f)(int x), int max){
	int tid = blockIdx.x + blockIdx.x * blockDim.x;
	if(tid < max) {
		out[tid] = f(a[tid]);
	}
}

__device__ int increment(int a){
	return a+1;
}
