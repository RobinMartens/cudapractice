
#include <hip/hip_runtime.h>
__global__ void vector_add(int *a, int *b, int *out, int max) {
	int tid = threadIdx.x; 
	if(tid < max) {
		out[tid] = a[tid] + b[tid];
	}
}
