#include <stdio.h>

#include "kernels.h"

#define N 10

int main(int argc, char **argv) {

	int a[N];
	int b[N];
	int out[N];

	// generate toydata
	for(int i=0; i < N; i++){
		a[i] = 2*i;
		b[i] = i*i;
	}

	int *a_;
	int *b_;
	int *out_;

	hipMalloc((void**)&a_, N * sizeof(int));
	hipMalloc((void**)&b_, N * sizeof(int));
	hipMalloc((void**)&out_, N * sizeof(int));

	hipMemcpy(a_, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_, b, N * sizeof(int), hipMemcpyHostToDevice);

	vector_add<<<N,1>>>(a_, b_, out_, N);

	hipMemcpy(out, out_, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(a_);
	hipFree(b_);
	hipFree(out_);

	for(int i=0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], out[i]);
	}

	return 0;
}
