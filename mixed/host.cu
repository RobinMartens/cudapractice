#include<stdio.h>

#define N 10

int main(int argc, char **argv) {

	int a[N];
	int b[N];
	int out[N];

	// generate toydata
	for(int i=0; i < N; i++){
		a[i] = 2*i;
		b[i] = i*i;
	}

	int *a_;
	int *b_;
	int *out_;

	hipMalloc((void**)&a_, N * sizeof(int));
	hipMalloc((void**)&b_, N * sizeof(int));
	hipMalloc((void**)&out_, N * sizeof(int));

	hipMemcpy(a, a_, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b, b_, N * sizeof(int), hipMemcpyHostToDevice);

	vector_add<<<N,1>>>(a_, b_, out_);

	hipMemcpy(out_, out, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(a_);
	hipFree(b_);
	hipFree(c_);

	return 0;
}
